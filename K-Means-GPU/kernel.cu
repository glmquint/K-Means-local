﻿#include "hip/hip_runtime.h"


#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <thread>
#include <vector>
#include <iostream>
#include <fstream>
#include <sstream>
#include <cstdlib>
#include <limits>
#include <string>
#include <chrono>
#include <math.h>
#include <time.h>
#include <cassert>

#define PRINT_CENTERS
#define PREALLOC_OPTIMIZE_OFF
#define STOPPING_ERROR 1e-2

using namespace std;
clock_t tic, toc;

int THREADS = 1;
bool CONVERGED = false;
int POINT_DIMENSION = 2;
int NUM_CLUSTERS = 2;
int DATASET_SIZE;
int THREADS_PER_BLOCK;

struct Point_s
{
	double coords[2];
};
typedef struct Point_s Point;

struct ClassedPoint_s
{
	Point p;
	int k;
};
typedef struct ClassedPoint_s ClassedPoint;

struct Centroid_s
{
	Point p;
	Point *sum;				// size(sum) == THREADS;
	int *partition_lengths; // size(partition_lengths) == THREADS;
};
typedef struct Centroid_s Centroid;

ClassedPoint *points;
Centroid *centroids;
ClassedPoint *d_points;
Point *d_centroids;
Point *d_centroids_sums;
int *d_centroids_plengths;
#ifdef PREALLOC_OPTIMIZE
Point* d_sum;
int* d_points_per_centroid;
#endif // PREALLOC_OPTIMIZE

// distance squared between 2 points
// root square is not necessarry for distance comparison
// and is removeed as optimization
__device__ double distance(Point &a, Point &b)
{
	double sum_of_squares = 0;
	double diff_coord;
	for (int i = 0; i < 2; ++i)
	{
		diff_coord = a.coords[i] - b.coords[i];
		sum_of_squares += (diff_coord * diff_coord);
	}
	return sum_of_squares;
}

double distanceCPU(Point &a, Point &b)
{
	double sum_of_squares = 0;
	double diff_coord;
	for (int i = 0; i < 2; ++i)
	{
		diff_coord = a.coords[i] - b.coords[i];
		sum_of_squares += (diff_coord * diff_coord);
	}
	return sum_of_squares;
}

#if __CUDA_ARCH__ < 600
__device__ double atomicAddDouble(double *address, double val)
{
	unsigned long long int *address_as_ull = (unsigned long long int *)address;
	unsigned long long int old = *address_as_ull, assumed;
	do
	{
		assumed = old;
		old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));

		// Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
	} while (assumed != old);
	return __longlong_as_double(old);
}
#endif

__global__ void worker(ClassedPoint *d_point, Point *d_centr, Point* d_centroids_sums, int * d_centroids_plengths, int dataset_size, 
int num_clusters, int partition_size, int num_threads
#ifdef PREALLOC_OPTIMIZE
, Point* d_sum, int* d_points_per_centroid
#endif // PREALLOC_OPTIMIZE
)
{
	double dist = 0;
	int best_k;
	double min_d;
	Point *sum;
	int *points_per_centroid;

#ifdef PREALLOC_OPTIMIZE
	sum = &d_sum[(blockDim.x * blockIdx.x + threadIdx.x)*num_clusters];
	points_per_centroid = &d_points_per_centroid[(blockDim.x * blockIdx.x + threadIdx.x)*num_clusters];
#else
	sum = new Point[num_clusters];
	points_per_centroid = new int[num_clusters];
#endif // PREALLOC_OPTIMIZE

	for (int j = 0; j < num_clusters; ++j)
	{
		for (int k = 0; k < 2; ++k)
		{
			sum[j].coords[k] = 0;
		}
		points_per_centroid[j] = 0;
	}

	int index = threadIdx.x + blockIdx.x * blockDim.x;
	for (int elem = 0; elem < partition_size; ++elem)
	{
		int partition_elem = partition_size * index + elem;
		if (partition_elem < dataset_size)
		{
			min_d = 1.7976931348623157e+308; // +inf
			best_k = -1;
			for (int i = 0; i < num_clusters; ++i)
			{
				dist = distance(d_point[partition_elem].p, d_centr[i]);
				/*
				if (dist < min_d)
				{
					min_d = dist;
					best_k = i;
				}
				*/
				best_k = i * (dist < min_d) + best_k * (dist >= min_d);
				min_d = dist * (dist < min_d) + min_d * (dist >= min_d);
			}
			d_point[partition_elem].k = best_k;
			for (int i = 0; i < 2; ++i)
			{
				sum[best_k].coords[i] += d_point[partition_elem].p.coords[i];
			}
			points_per_centroid[best_k]++;
		}
	}
	//printf("%d) %f\n", index, sum[0].coords[0]);
	for (int i = 0; i < num_clusters; ++i)
	{
		for (int j = 0; j < 2; ++j)
		{
			d_centroids_sums[i * num_threads + index].coords[j] = sum[i].coords[j];
		}
		d_centroids_plengths[i * num_threads + index] = points_per_centroid[i];
	}
	//printf("%d} %f\n", index, d_centroids_sums[0 + index].coords[0]);
	// ok????
#ifndef PREALLOC_OPTIMIZE
	delete[] sum;
	delete[] points_per_centroid;
#endif // !PREALLOC_OPTIMIZE

}

void updateCenters()
{
	double max_err = numeric_limits<double>::min();
	for (int i = 0; i < NUM_CLUSTERS; ++i)
	{
		Point point_sum = {};
		// point_sum.coords = new double[POINT_DIMENSION];
		for (int k = 0; k < POINT_DIMENSION; ++k)
		{
			point_sum.coords[k] = 0;
		}
		int sum_of_lengths = 0;

		for (int j = 0; j < THREADS; j++)
		{
			for (int k = 0; k < POINT_DIMENSION; ++k)
			{
				point_sum.coords[k] += centroids[i].sum[j].coords[k];
			}
			// point_sum.coords[k] = centroids[i].sum.coords[k];
			sum_of_lengths += centroids[i].partition_lengths[j];
		}

		//double point_sum_square_norm = 0;
		for (int j = 0; j < POINT_DIMENSION; ++j)
		{
			point_sum.coords[j] /= sum_of_lengths; // new centroid
			//point_sum_square_norm += (point_sum.coords[j] * point_sum.coords[j]);
		}
		double dist = distanceCPU(centroids[i].p, point_sum);
		for (int j = 0; j < POINT_DIMENSION; ++j)
		{
			centroids[i].p.coords[j] = point_sum.coords[j];
		}

		double error = sqrt(dist); // point_sum_square_norm;
		if (error > max_err)
		{
			max_err = error;
		}
#ifdef PRINT_CENTERS
		printf("centroid %d (%f:%f) with %d elements (error: %f)\n", i, centroids[i].p.coords[0], centroids[i].p.coords[1], sum_of_lengths, error);
#endif
	}
#ifdef PRINT_CENTERS
	printf("==================================================\n");
#endif
	CONVERGED = (max_err < STOPPING_ERROR);
	// CONVERGED = true;
}

void performRounds(dim3 grid, dim3 block, int partition_size)
{
	int round = 0;
	while (!CONVERGED)
	{
		// for (int thread_i = 0; thread_i < THREADS; ++thread_i)
		// {
		// 	threads[thread_i] = thread(worker, first_points[thread_i], partition_lengths[thread_i], thread_i);
		// }
		// cudaMemcpy(d_centroids, centroids, NUM_CLUSTERS * sizeof(Centroid), cudaMemcpyHostToDevice);
		hipError_t cerr;
		for (int i = 0; i < NUM_CLUSTERS; ++i) {
			cerr = hipMemcpy(&d_centroids[i], &centroids[i], sizeof(Point), hipMemcpyHostToDevice);
		}
		assert(cerr == hipSuccess);
		worker<<<grid, block>>>(d_points, d_centroids, d_centroids_sums, d_centroids_plengths, DATASET_SIZE, NUM_CLUSTERS, partition_size, THREADS
#ifdef PREALLOC_OPTIMIZE
		, d_sum, d_points_per_centroid
#endif // PREALLOC_OPTIMIZE
		);
		hipDeviceSynchronize();
		
		for (int i = 0; i < NUM_CLUSTERS; ++i)
		{
			cerr = hipMemcpy(centroids[i].sum, &d_centroids_sums[i*THREADS], THREADS * sizeof(Point), hipMemcpyDeviceToHost);
			assert(cerr == hipSuccess);
			cerr = hipMemcpy(centroids[i].partition_lengths, &d_centroids_plengths[i*THREADS], THREADS * sizeof(int), hipMemcpyDeviceToHost);
			assert(cerr == hipSuccess);
		}
		/*
		int count = 0;
		for (int i = 0; i < NUM_CLUSTERS; ++i) {
			count += centroids[i].partition_lengths;
		}
		assert(count == DATASET_SIZE, "didn't count enough points\n");
		*/
		updateCenters();
		round++;
		// printf("%f\n", round, elapsed);
	}
#ifdef PRINT_CENTERS
	 printf("took %d rounds\n", round);
#endif
}

void setupRandomCentroids()
{

	srand(69420);
	for (int i = 0; i < NUM_CLUSTERS; ++i)
	{
		int random_index = rand() % (DATASET_SIZE);
		for (int coord = 0; coord < POINT_DIMENSION; coord++)
		{
			centroids[i].p.coords[coord] = points[random_index].p.coords[coord];
		}
		for (int j = 0; j < THREADS; j++)
		{
			for (int k = 0; k < POINT_DIMENSION; ++k)
			{
				centroids[i].sum[j].coords[k] = 0;
			}
			centroids[i].partition_lengths[j] = 0;
		}
	}
}

void generateRandomCentroids()
{
	centroids = new Centroid[NUM_CLUSTERS];
	for (int i = 0; i < NUM_CLUSTERS; ++i){
		//Centroid* c = new Centroid;
		//c->p = *new Point;
		//c->p.coords = new double[POINT_DIMENSION];
		centroids[i].sum = new Point[THREADS];
		centroids[i].partition_lengths = new int[THREADS];
		//c->sum[j].coords = new double[POINT_DIMENSION];
		for (int j = 0; j < THREADS; j++){
			for (int k = 0; k < POINT_DIMENSION; ++k){
				centroids[i].sum[j].coords[k] = 0;
			}
			centroids[i].partition_lengths[j] = 0;
		}
	}
}

void deserializePoints(char *intput_file)
{
	ifstream infile;
	infile.open(intput_file, ios::in | ios::binary);
	if (infile.fail())
	{
		cout << "can't find file " << intput_file << endl;
		exit(1);
	}
	infile.read((char *)(&DATASET_SIZE), sizeof(DATASET_SIZE));
	points = new ClassedPoint[DATASET_SIZE];
	infile.read((char *)(&POINT_DIMENSION), sizeof(POINT_DIMENSION));
	for (int i = 0; i < DATASET_SIZE; i++)
	{
		// points[i].p.coords = new double[POINT_DIMENSION];
		points[i].k = -1;
		for (int j = 0; j < POINT_DIMENSION; ++j)
		{
			infile.read((char *)(&points[i].p.coords[j]), sizeof(double));
		}
	}
	infile.close();
}

int main(int argc, char **argv)
{
	if (argc < 5)
	{
		printf("[USAGE]: %s dataset.serialized num_clusters num_threads threads_per_block\n", argv[0]);
		exit(1);
	}
	NUM_CLUSTERS = stoi(argv[2]);
	centroids = new Centroid[NUM_CLUSTERS];

	THREADS = stoi(argv[3]);
	THREADS_PER_BLOCK = stoi(argv[4]);

	deserializePoints(argv[1]);
	generateRandomCentroids();

	int num_blocks = THREADS / THREADS_PER_BLOCK;
	if (THREADS % THREADS_PER_BLOCK)
		num_blocks++;

	int partition_size;
	if (DATASET_SIZE % THREADS == 0)
	{
		partition_size = DATASET_SIZE / THREADS;
	}
	else
	{
		partition_size = DATASET_SIZE / (THREADS - 1);
	}

	dim3 grid(num_blocks, 1, 1);
	dim3 block(THREADS_PER_BLOCK, 1, 1);
	
	hipMalloc((void **) &d_points, DATASET_SIZE * sizeof(ClassedPoint));
	hipMalloc((void **) &d_centroids, NUM_CLUSTERS * sizeof(Point));
	hipMalloc((void **) &d_centroids_sums, NUM_CLUSTERS * THREADS * sizeof(Point));
	hipMalloc((void **) &d_centroids_plengths, NUM_CLUSTERS * THREADS * sizeof(int));

#ifdef PREALLOC_OPTIMIZE
	cudaMalloc((void **) &d_sum, NUM_CLUSTERS * THREADS * sizeof(Point));
	cudaMalloc((void **) &d_points_per_centroid, NUM_CLUSTERS * THREADS * sizeof(int));
#endif // PREALLOC_OPTIMIZE

	// must copy to device at each repetition
	// do it once for every repetition
	hipError_t cerr;
	clock_t ds_tic = clock();
	cerr = hipMemcpy(d_points, points, DATASET_SIZE * sizeof(ClassedPoint), hipMemcpyHostToDevice);
	clock_t ds_toc = clock();
	assert(cerr == hipSuccess);
	for (int rep = 0; rep < 30; rep++)
	{
		setupRandomCentroids();
		for (int i = 0; i < DATASET_SIZE; i++)
		{
			points[i].k = -1;
		}

		CONVERGED = false;
		// copy from host to device
		clock_t tic = clock();
		clock_t intermidiate_clock = clock();
		performRounds(grid, block, partition_size);
		clock_t toc = clock();
#ifdef PRINT_CENTERS
		printf("execution time: %f (dataset load %f)\n", (double)(toc - tic) / CLOCKS_PER_SEC, (double)(ds_toc - ds_tic) / CLOCKS_PER_SEC);
#else
		printf("Ok: %f\n", (double)(toc - intermidiate_clock) / CLOCKS_PER_SEC);
#endif
		/*
		for (int i = 0; i < NUM_CLUSTERS; ++i) {
			printf("(%f %f)\n", centroids[i].p.coords[0], centroids[i].p.coords[1]);
		}
		*/
		/*
		auto start = std::chrono::high_resolution_clock::now();
		performRounds(threads, first_points, partition_lengths);
		auto end = std::chrono::high_resolution_clock::now();
		std::chrono::duration<double> diff = end - start;
		printf("%f\n", diff.count());
		*/

#ifdef PRINT_CENTERS
		printf("/------------begin centroids-------------\\\n");
		for (int i = 0; i < NUM_CLUSTERS; i++)
		{
			printf("Centro %d : ", i);
			for (int j = 0; j < POINT_DIMENSION; j++)
			{
				printf("%f ", centroids[i].p.coords[j]);
			}
			printf("\n");
		}
		printf("\\------------end centroids---------------/\n");
#endif
	}
	hipFree(d_points);
	hipFree(d_centroids);
	hipFree(d_centroids_sums);
	hipFree(d_centroids_plengths);

	return 0;
}
